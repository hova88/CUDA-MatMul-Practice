#include "hip/hip_runtime.h"
/*
 * Mark Harris
 * NVIDIA Developer Technology
 */
 #include <stdio.h>
 #include "../parser.h"

 template<unsigned int blocksize>
 __device__ void warpReduce(volatile float* sdata , int tid) {
     if (blocksize >= 64) sdata[tid] += sdata[tid + 32];
     if (blocksize >= 32) sdata[tid] += sdata[tid + 16];
     if (blocksize >= 16) sdata[tid] += sdata[tid +  8];
     if (blocksize >=  8) sdata[tid] += sdata[tid +  4];
     if (blocksize >=  4) sdata[tid] += sdata[tid +  2];
     if (blocksize >=  2) sdata[tid] += sdata[tid +  1];
 }


__global__ void
sequential_address_unrolling_kernel(float* g_odata, float* g_idata)
{
    extern __shared__ float sdata[];
    
    // each thread loads one element from global to shared memory
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * (blockDim.x * 2 ) + threadIdx.x;
    sdata[tid] = g_idata[i] + g_idata[i + blockDim.x];
    __syncthreads();

    // do reduction in shared memory
    for (unsigned int s = blockDim.x /2 ; s > 32 ; s>>=1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }

    if (tid < 32) warpReduce<2>(sdata , tid);
    // write result for this block to global memory
    if (tid == 0 ) g_odata[blockIdx.x] = sdata[0];

}

  
void parser::reduce_sequnroll( matrix& C) {
    
	float* dev_a;
	hipMalloc(&dev_a, A.row * A.col * sizeof(float));
	hipMemcpy(dev_a, A.elements,  A.row * A.col * sizeof(float), hipMemcpyHostToDevice);
    
    float* dev_c;
    hipMalloc(&dev_c, C.row  * sizeof(float));
    
    sequential_address_unrolling_kernel<<< 3 , 2 , 32 * sizeof(float)>>>(dev_c, dev_a);
    hipDeviceSynchronize();
    
    hipMemcpy(C.elements, dev_c, C.row * sizeof(float), hipMemcpyDeviceToHost);
    hipFree(dev_a);
    hipFree(dev_c);
    return;
}