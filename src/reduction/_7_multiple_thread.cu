#include "hip/hip_runtime.h"
/*
 * Mark Harris
 * NVIDIA Developer Technology
 */
 #include <stdio.h>
 #include "../parser.h"
template<unsigned int blocksize>
__device__ void warpReduce(volatile float* sdata , int tid) {
    if (blocksize >= 64) sdata[tid] += sdata[tid + 32];
    if (blocksize >= 32) sdata[tid] += sdata[tid + 16];
    if (blocksize >= 16) sdata[tid] += sdata[tid +  8];
    if (blocksize >=  8) sdata[tid] += sdata[tid +  4];
    if (blocksize >=  4) sdata[tid] += sdata[tid +  2];
    if (blocksize >=  2) sdata[tid] += sdata[tid +  1];
}

template<unsigned int blocksize>
__global__ void
multiple_thread_kernel(float* g_odata, float* g_idata)
{
    extern __shared__ float sdata[];
    
    // each thread loads one element from global to shared memory
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * ( blocksize * 2 ) + threadIdx.x;
    unsigned int gridsize = blocksize * 2 * gridDim.x;
    unsigned int n = 12; // number of elements
    sdata[tid] = 0;
    
    while (i < n) {
        sdata[tid] += g_idata[i] + g_idata[i+blocksize];
        i += gridsize;
    }
    __syncthreads();

    // do reduction in shared memory
    if (blocksize >= 512) {
        if (tid < 256) { sdata[tid] += sdata[tid + 256] ;} __syncthreads();}
    if (blocksize >= 256) {
        if (tid < 128) { sdata[tid] += sdata[tid + 128] ;} __syncthreads();}
    if (blocksize >= 128) {
        if (tid < 64) { sdata[tid] += sdata[tid +   64] ;} __syncthreads();}
    
    
    if (tid < 32) warpReduce<blocksize>(sdata , tid);
    // write result for this block to global memory
    if (tid == 0 ) g_odata[blockIdx.x] = sdata[0];

}

  
void parser::multiple_thread( matrix& C) {
    
	float* dev_a;
	hipMalloc(&dev_a, A.row * A.col * sizeof(float));
	hipMemcpy(dev_a, A.elements,  A.row * A.col * sizeof(float), hipMemcpyHostToDevice);
    
    float* dev_c;
    hipMalloc(&dev_c, C.row  * sizeof(float));
    // switch (threads)
    multiple_thread_kernel<2><<< 3 , 2 , 32 * sizeof(float)>>>(dev_c, dev_a);
    hipDeviceSynchronize();
    
    hipMemcpy(C.elements, dev_c, C.row * sizeof(float), hipMemcpyDeviceToHost);
    hipFree(dev_a);
    hipFree(dev_c);
    return;
}