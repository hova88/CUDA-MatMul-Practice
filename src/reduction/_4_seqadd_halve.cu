#include "hip/hip_runtime.h"
/*
 * Mark Harris
 * NVIDIA Developer Technology
 */
 #include <stdio.h>
 #include "../parser.h"

__global__ void
sequential_address_halve_kernel(float* g_odata, float* g_idata)
{
    extern __shared__ float sdata[];
    
    // each thread loads one element from global to shared memory
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * (blockDim.x * 2 ) + threadIdx.x;
    sdata[tid] = g_idata[i] + g_idata[i + blockDim.x];
    __syncthreads();

    // do reduction in shared memory
    for (unsigned int s = blockDim.x /2 ; s > 0 ; s>>=1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }
    // write result for this block to global memory
    if (tid == 0 ) g_odata[blockIdx.x] = sdata[0];

}

  
void parser::reduce_seqhalve( matrix& C) {
    
	float* dev_a;
	hipMalloc(&dev_a, A.row * A.col * sizeof(float));
	hipMemcpy(dev_a, A.elements,  A.row * A.col * sizeof(float), hipMemcpyHostToDevice);
    
    float* dev_c;
    hipMalloc(&dev_c, C.row  * sizeof(float));
    
    sequential_address_halve_kernel<<< 3 , 2 , 32 * sizeof(float)>>>(dev_c, dev_a);
    hipDeviceSynchronize();
    
    hipMemcpy(C.elements, dev_c, C.row * sizeof(float), hipMemcpyDeviceToHost);
    hipFree(dev_a);
    hipFree(dev_c);
    return;
}