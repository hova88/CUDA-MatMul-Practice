#include <cmath>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <chrono>

#include <hip/hip_runtime.h>

#include "./src/utils.cuh"
#include "./src/1_naive.cuh"
#include "./src/2_global_memory_coalescing.cuh"
#include "./src/3_shared_memeory.cuh"
#include "./src/4_blocktiling_1D.cuh"


/**
 * O(N^3) naive GEMM implementation (inner product)
 * 
 * for large square matrices where M=N=K, 
 * the number of math operations in a product of matrices is O(N3) 
 * while the amount of data needed is O(N2), 
 * yielding a compute intensity on the order of N.
 */
void matmul_cpu(const float* A, const float* B, float* C, float* C_result,
                      int M, int N, int K, float alpha, float beta) {
  for (int i = 0; i < M; i++) {
    for (int j = 0; j < N; j++){
      float sum = 0.0f;
      for (int k = 0; k < K; k++) {
        sum += A[i * K + k] * B[k * N + j];
      }
      C_result[i * N + j] = alpha * sum + beta * C[i * N + j];
    }
  }
}

void launch_and_verify_kernel(const char* kernel_info,
                              void (*kernel_launcher)(const float*, const float*, float*, int, int, int, float, float),
                              float* d_A, float* d_B, float* d_C_ref,
                              float* C, float* C_ref, float* C_after_gemm_cpu,
                              int M, int N, int K,
                              float alpha, float beta) {
  checkCudaErrors(hipMemcpy(d_C_ref, C, M * N * sizeof(float), hipMemcpyHostToDevice));
  kernel_launcher(d_A, d_B, d_C_ref, M, N, K, alpha, beta);
  checkCudaErrors(hipDeviceSynchronize());
  checkCudaErrors(hipMemcpy(C_ref, d_C_ref, M * N * sizeof(float), hipMemcpyDeviceToHost));
  if (verify_matrix(C_after_gemm_cpu, C_ref, M * N)) {
    printf("[%s]: Correct!\n", kernel_info);
  } else {
    printf("[%s]: Wrong!\n", kernel_info);
  }
}

void time_and_print_kernel(const char* kernel_info,
                           void (*kernel_launcher)(const float*, const float*, float*, int, int, int, float, float),
                           float* d_A, float* d_B, float* d_C, int M, int N, int K,
                           float alpha, float beta, int repeat_times) {
  hipEvent_t beg, end;
  float elapsed_time;

  hipEventCreate(&beg);
  hipEventCreate(&end);

  hipEventRecord(beg);
  for (int i = 0; i < repeat_times; i++) {
    kernel_launcher(d_A, d_B, d_C, M, N, K, alpha, beta);
  }
  hipEventRecord(end);
  hipEventSynchronize(end);
  hipEventElapsedTime(&elapsed_time, beg, end);

  printf("%s: %f ms\n", kernel_info, elapsed_time / repeat_times);

  hipEventDestroy(beg);
  hipEventDestroy(end);
}

int main(int argc, char** argv) {

  CudaDeviceInfo();

  // Using cudaEvent for gpu stream timing, cudaEvent is equivalent to
  // publishing event tasks in the target stream
  int repeat_times = 1000;

  int M = 1024, N = 1024, K = 1024;
  float alpha = 1.0f, beta = 1.0f; // GEMM input parameters, C=α*AB+β*C

  float *A, *B, *C, *C_ref, *C_after_gemm_cpu; // host matrices
  float *d_A, *d_B, *d_C, *d_C_ref; // device matrices

  // allocate host memory
  A = (float*)malloc(M * K * sizeof(float));
  B = (float*)malloc(K * N * sizeof(float));
  C = (float*)malloc(M * N * sizeof(float));
  C_ref = (float*)malloc(M * N * sizeof(float));
  C_after_gemm_cpu = (float*)malloc(M * N * sizeof(float));

  // allocate device memory
  checkCudaErrors(hipMalloc((void**)&d_A, M * K * sizeof(float)));
  checkCudaErrors(hipMalloc((void**)&d_B, K * N * sizeof(float)));
  checkCudaErrors(hipMalloc((void**)&d_C, M * N * sizeof(float)));
  checkCudaErrors(hipMalloc((void**)&d_C_ref, M * N * sizeof(float)));

  // initialize host matrices
  random_init_matrix(A, M * K);
  random_init_matrix(B, K * N);
  random_init_matrix(C, M * N);


  // copy host matrices to device
  checkCudaErrors(hipMemcpy(d_A, A, M*K*sizeof(float), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_B, B, K*N*sizeof(float), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_C, C, M*N*sizeof(float), hipMemcpyHostToDevice));

  // verify correctness of naive GEMM
  matmul_cpu(A, B, C, C_after_gemm_cpu, M, N, K, alpha, beta);

  launch_and_verify_kernel("kernel_1(Naive GEMM)", 
                           matmul_naive_launcher, 
                           d_A, d_B, d_C_ref, 
                           C, C_ref, C_after_gemm_cpu, M, N, K, alpha, beta);

  launch_and_verify_kernel("kernel_2(Global Memory coalesce)", 
                           matmul_naive_launcher, 
                           d_A, d_B, d_C_ref, 
                           C, C_ref, C_after_gemm_cpu, M, N, K, alpha, beta);

  launch_and_verify_kernel("kernel_3(Shared Memory + Global Memory Coalesce)", 
                           matmul_naive_launcher, 
                           d_A, d_B, d_C_ref, 
                           C, C_ref, C_after_gemm_cpu, M, N, K, alpha, beta);

  launch_and_verify_kernel("kernel_4(Shared Memory + Global Memory Coalesce + Block Tiling 1D)", 
                           matmul_naive_launcher, 
                           d_A, d_B, d_C_ref, 
                           C, C_ref, C_after_gemm_cpu, M, N, K, alpha, beta);

  // warm up
  printf("Warming up...\n");
  for (int i = 0; i < repeat_times; i++) {
    matmul_naive_launcher(d_A, d_B, d_C, M, N, K, alpha, beta);
    matmul_global_mem_coalesce_launcher(d_A, d_B, d_C, M, N, K, alpha, beta);
    matmul_shared_mem_block_launcher(d_A, d_B, d_C, M, N, K, alpha, beta);
  }

  // let's time it
  printf(" --- GEMM Performance ---\n");
  time_and_print_kernel("1. Naive GEMM", 
                        matmul_naive_launcher, 
                        d_A, d_B, d_C, M, N, K, alpha, beta, repeat_times);

  time_and_print_kernel("2. Global Memory coalesce", 
                        matmul_global_mem_coalesce_launcher, 
                        d_A, d_B, d_C, M, N, K, alpha, beta, repeat_times);

  time_and_print_kernel("3. Shared Memory + Global Memory Coalesce", 
                        matmul_shared_mem_block_launcher, 
                        d_A, d_B, d_C, M, N, K, alpha, beta, repeat_times);

  time_and_print_kernel("4. Shared Memory + Global Memory Coalesce + Block Tiling 1D", 
                        matmul_smem_blocktiling1d_launcher, 
                        d_A, d_B, d_C, M, N, K, alpha, beta, repeat_times);

  // Free up CPU and GPU space
  free(A);
  free(B);
  free(C);
  free(C_ref);
  free(C_after_gemm_cpu);
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
  hipFree(d_C_ref);
	return 0;
}
